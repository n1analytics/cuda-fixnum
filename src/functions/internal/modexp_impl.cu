#pragma once
#include <hip/hip_runtime.h>


namespace internal
{
    /*
     * Return floor(log2(x)). In particular, if x = 2^b, return b.
     */
    __device__
    constexpr unsigned
    floorlog2(unsigned x) {
        return x == 1 ? 0 : 1 + floorlog2(x >> 1);
    }

    /*
     * The following function gives a reasonable choice of WINDOW_SIZE in the k-ary
     * modular exponentiation method for a fixnum of B = 2^b bytes.
     *
     * The origin of the table is as follows. The expected number of multiplications
     * for the k-ary method with n-bit exponent and d-bit window is given by
     *
     *   T(n, d) = 2^d - 2 + n - d + (n/d - 1)*(1 - 2^-d)
     *
     * (see Koç, C. K., 1995, "Analysis of Sliding Window Techniques for
     * Exponentiation", Equation 1). The following GP script calculates the values
     * of n at which the window size should increase (maximum n = 65536):
     *
     *   ? T(n,d) = 2^d - 2 + n - d + (n/d - 1) * (1 - 2^-d);
     *   ? M = [ vecsort([[n, d, T(n, d)*1.] | d <- [1 .. 16]], 3)[1][2] | n <- [1 .. 65536] ];
     *   ? maxd = M[65536]
     *   10
     *   ? [[d, vecmin([n | n <- [1 .. 65536], M[n] == d])] | d <- [1 .. maxd]]
     *   [[1, 1], [2, 7], [3, 35], [4, 122], [5, 369], [6, 1044], [7, 2823], [8, 7371], [9, 18726], [10, 46490]]
     *
     * Table entry i is the window size for a fixnum of 8*(2^i) bits (e.g. 512 =
     * 8*2^6 bits falls between 369 and 1044, so the window size is that of the
     * smaller, 369, so 5 is in place i = 6).
     */
    // NB: For some reason we're not allowed to put this table in the definition
    // of bytes_to_window_size().
    constexpr int BYTES_TO_WINDOW_SIZE_TABLE[] = {
       -1,
       -1, //bytes bits
        2, // 2^2    32
        3, // 2^3    64
        4, // 2^4   128
        4, // 2^5   256
        5, // 2^6   512
        5, // 2^7  1024
        6, // 2^8  2048
        7, // 2^9  4096
        8, //2^10  8192
        8, //2^11 16384
        9, //2^12 32768
        10,//2^13 65536
    };

    __device__
    constexpr int
    bytes_to_window_size(unsigned bytes) {
        return BYTES_TO_WINDOW_SIZE_TABLE[floorlog2(bytes)];
    }
}
